
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
// Kernel Needs to be modified to make sure the parallalization happens 
// For that I need thread id and the number of threads in that block
__global__
void add(int n, float* x, float* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = 0; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20;
    float* x, * y;

    // Allocate Unified Memory � accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    std::cout << "Blocks: " << numBlocks << std::endl;
    std::cout << "Threads: " << blockSize << std::endl;

    // Run kernel on 1M elements on the GPU
    add <<< numBlocks, blockSize >>> (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}